#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <omp.h>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#define NUM_ANTS 50
#define NUM_ITERATIONS 100
#define ALPHA 1.0
#define BETA 2.0
#define RHO 0.2
#define Q 1.0
#define GRID 10
#define BLOCK 5

struct Point {
    int id;
    float x;
    float y;
};

class TSPReader {
  private:
    std::string name;
    std::string comment;
    std::string type;
    int dimension;
    std::string edge_weight_type;
    std::vector<Point> points;

  public:
    bool readFile(const std::string &filename)
    {
        std::ifstream file(filename);
        if (!file.is_open()) {
            std::cerr << "Error: Could not open file " << filename << std::endl;
            return false;
        }

        std::string line;
        bool reading_coords = false;

        while (std::getline(file, line)) {
            if (reading_coords) {
                std::istringstream iss(line);
                Point p;
                if (iss >> p.id >> p.x >> p.y) {
                    points.push_back(p);
                }
            }
            else {
                if (line.find("NAME") != std::string::npos) {
                    name = line.substr(line.find(":") + 1);
                    name = name.substr(name.find_first_not_of(" \t"));
                }
                else if (line.find("COMMENT") != std::string::npos) {
                    comment = line.substr(line.find(":") + 1);
                }
                else if (line.find("TYPE") != std::string::npos) {
                    type = line.substr(line.find(":") + 1);
                }
                else if (line.find("DIMENSION") != std::string::npos) {
                    dimension = std::stoi(line.substr(line.find(":") + 1));
                }
                else if (line.find("EDGE_WEIGHT_TYPE") != std::string::npos) {
                    edge_weight_type = line.substr(line.find(":") + 1);
                }
                else if (line.find("NODE_COORD_SECTION") != std::string::npos) {
                    reading_coords = true;
                }
            }
        }

        file.close();
        return true;
    }
    void printData() const
    {
        std::cout << "Name: " << name << std::endl;
        std::cout << "Comment: " << comment << std::endl;
        std::cout << "Type: " << type << std::endl;
        std::cout << "Dimension: " << dimension << std::endl;
        std::cout << "Edge Weight Type: " << edge_weight_type << std::endl;
        std::cout << "\nCoordinates:" << std::endl;
        for (const auto &point : points) {
            std::cout << "Point " << point.id << ": (" << point.x << ", " << point.y << ")" << std::endl;
        }
    }
    const std::vector<Point> &getPoints() const { return points; }
};

class TSPSolver {
  protected:
    // data
    std::vector<Point> points;
    float totalDistance;

    // functions
    float calculateDistance(const Point &p1, const Point &p2) const
    {
        float dx = p1.x - p2.x;
        float dy = p1.y - p2.y;
        return std::sqrt(dx * dx + dy * dy);
    }
    float calculateTourDistance(const std::vector<int> &tour) const
    {
        float distance = 0;
        for (size_t i = 0; i < tour.size() - 1; ++i) {
            distance += calculateDistance(points[tour[i]], points[tour[i + 1]]);
        }
        distance += calculateDistance(points[tour.back()], points[tour.front()]);
        return distance;
    }

  public:
    TSPSolver(const std::vector<Point> &pts) : points(pts), totalDistance(std::numeric_limits<float>::infinity()) {}
    float getDistance() { return totalDistance; }
    // void printTour() const;
};

__global__ void setup_kernel(hiprandState *state, unsigned long t)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(t, idx, 0, &state[idx]);
}

__device__ float calculateDistanceCuda(const Point &p1, const Point &p2)
{
    float dx = p1.x - p2.x;
    float dy = p1.y - p2.y;
    return sqrtf(dx * dx + dy * dy);
}

__device__ int selectNextCityCuda(hiprandState *my_curandstate, int idx, float *d_pheromones, const Point *points,
                                  const int numCities, int current, bool *visited)
{
    float *prob = new float[numCities], total_prob = 0.0;
    memset(prob, 0, numCities * sizeof(float));

    for (int nextCity = 0; nextCity < numCities; nextCity++) {
        if (!visited[nextCity]) {
            prob[nextCity] = powf(d_pheromones[current * numCities + nextCity], ALPHA) *
                             powf(1.0 / calculateDistanceCuda(points[current], points[nextCity]), BETA);
            total_prob += prob[nextCity];
        }
    }

    float r = hiprand_uniform(&(my_curandstate[idx])) * total_prob;
    for (int i = 0; i < numCities; i++) {
        if (!visited[i]) {
            r -= prob[i];
            if (r <= 0) {
                delete[] prob;
                return i;
            }
        }
    }

    for (int i = 0; i < numCities; i++) {
        if (!visited[i]) {
            delete[] prob;
            return i;
        }
    }

    return -1;
}

__device__ void _2OptCuda(int *tour, const Point *points, const int numCities)
{
    bool improved = true;
    while (improved) {
        improved = false;
        for (int i = 0; i < numCities - 1; i++) {
            for (int j = i + 1; j < numCities; j++) {
                float beforeDistance = calculateDistanceCuda(points[tour[i]], points[tour[i + 1]]) +
                                       calculateDistanceCuda(points[tour[j]], points[tour[(j + 1)]]);

                float afterDistance = calculateDistanceCuda(points[tour[i]], points[tour[j]]) +
                                      calculateDistanceCuda(points[tour[i + 1]], points[tour[(j + 1)]]);

                if (afterDistance < beforeDistance) {
                    int start = i + 1, end = j;
                    while (start < end) {
                        int temp = tour[start];
                        tour[start] = tour[end];
                        tour[end] = temp;
                        start++;
                        end--;
                    }
                    improved = true;
                }
            }
        }
    }
}

__device__ float calculateTourDistanceCuda(int *tour, const Point *points, const int numCities)
{
    float distance = 0;
    for (int i = 0; i < numCities - 1; i++) {
        distance += calculateDistanceCuda(points[tour[i]], points[tour[i + 1]]);
    }
    distance += calculateDistanceCuda(points[tour[numCities - 1]], points[tour[0]]);
    return distance;
}

__global__ void contructSolutionCuda(hiprandState *my_curandstate, int *ants_tour, float *d_pheromones,
                                     const Point *points, const int numCities, float *bestDist, int *lock)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    int *d_ant = new int[numCities];
    for (int i = 0; i < numCities; i++) {
        d_ant[i] = -1;
    }

    float myrandf = hiprand_uniform(&(my_curandstate[idx]));
    myrandf *= ((float)(numCities - 1) + 0.999999);
    int current = (int)truncf(myrandf);

    d_ant[0] = current;
    bool *visited = new bool[numCities];
    memset(visited, false, numCities * sizeof(bool));
    visited[current] = true;

    for (int i = 1; i < numCities; i++) {
        int next = selectNextCityCuda(my_curandstate, idx, d_pheromones, points, numCities, current, visited);
        d_ant[i] = next;
        visited[next] = true;
        current = next;
    }

    _2OptCuda(d_ant, points, numCities);
    float tour_len = calculateTourDistanceCuda(d_ant, points, numCities);

    bool leave = true;
    while (leave) {
        if (atomicCAS(lock, 0, 1) == 0) {
            if (tour_len < *bestDist) {
                *bestDist = tour_len;

                // copy ant tour to global memory
                for (int i = 0; i < numCities; i++) {
                    ants_tour[i] = d_ant[i];
                }
            }

            leave = false;
            atomicExch(lock, 0);
        }
        // break;
    }

    // free
    delete[] visited;
    delete[] d_ant;
}

class ACOCUDA : public TSPSolver {
  protected:
    // data
    float tau_max, tau_min;
    int numCities;
    // std::vector<std::vector<float>> pheromones;
    std::vector<float> pheromones;
    std::vector<int> bestTour;
    float globalBestDistance = std::numeric_limits<float>::infinity();

    // rng
    std::random_device rd;
    std::mt19937 gen;
    std::uniform_real_distribution<float> uniform_dist;

    // functions
    void updatePheromones(const std::vector<int> &allTours)
    {
        // evaporation
        for (int i = 0; i < numCities * numCities; i++) {
            pheromones[i] *= (1.0 - RHO);
        }

        int city1, city2;
        float deposit = 1.0 / globalBestDistance;
        for (int i = 0; i < numCities - 1; i++) {
            city1 = allTours[i];
            city2 = allTours[i + 1];
            pheromones[city1 * numCities + city2] += deposit;
            pheromones[city2 * numCities + city1] += deposit;
            pheromones[city1 * numCities + city2] = std::clamp(pheromones[city1 * numCities + city2], tau_min, tau_max);
            pheromones[city2 * numCities + city1] = std::clamp(pheromones[city2 * numCities + city1], tau_min, tau_max);
        }

        // Add return to start
        city1 = allTours[numCities - 1];
        city2 = allTours[0];
        pheromones[city1 * numCities + city2] += deposit;
        pheromones[city2 * numCities + city1] += deposit;
        pheromones[city1 * numCities + city2] = std::clamp(pheromones[city1 * numCities + city2], tau_min, tau_max);
        pheromones[city2 * numCities + city1] = std::clamp(pheromones[city2 * numCities + city1], tau_min, tau_max);
    }

  public:
    ACOCUDA(const std::vector<Point> &pts) : TSPSolver(pts)
    {
        numCities = points.size();
        pheromones.resize(numCities * numCities, 1.0);
    }

    // Main solving method
    void solve()
    {
        hiprandState *d_state;
        time_t t;
        time(&t);
        hipMalloc(&d_state, NUM_ANTS * sizeof(hiprandState));
        setup_kernel<<<GRID, BLOCK>>>(d_state, t);
        hipDeviceSynchronize();

        int *ants_tour;
        hipMalloc(&ants_tour, numCities * sizeof(int));
        hipMemset(ants_tour, -1, numCities * sizeof(int));

        float *d_pheromones;
        hipMalloc(&d_pheromones, numCities * numCities * sizeof(float));
        hipMemcpy(d_pheromones, pheromones.data(), numCities * numCities * sizeof(float), hipMemcpyHostToDevice);

        Point *d_points;
        hipMalloc(&d_points, numCities * sizeof(Point));
        hipMemcpy(d_points, points.data(), numCities * sizeof(Point), hipMemcpyHostToDevice);

        std::vector<int> tour(numCities);
        float tourLength = globalBestDistance;
        for (int iter = 0; iter < NUM_ITERATIONS; iter++) {
            float *cudaBestDist;
            hipMalloc(&cudaBestDist, sizeof(float));
            hipMemcpy(cudaBestDist, &globalBestDistance, sizeof(float), hipMemcpyHostToDevice);

            int *lock;
            hipMalloc(&lock, sizeof(int));
            hipMemset(lock, 0, sizeof(int));

            contructSolutionCuda<<<GRID, BLOCK>>>(d_state, ants_tour, d_pheromones, d_points, numCities, cudaBestDist,
                                                  lock);
            hipDeviceSynchronize();

            hipMemcpy(tour.data(), ants_tour, numCities * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(pheromones.data(), d_pheromones, numCities * numCities * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&tourLength, cudaBestDist, sizeof(float), hipMemcpyDeviceToHost);

            if (tourLength < globalBestDistance) {
                bestTour = tour;
                globalBestDistance = tourLength;
            }

            tau_max = 1.0 / (RHO * globalBestDistance);
            tau_min = tau_max * (1.0 - std::pow(0.05, 1.0 / numCities)) /
                      ((numCities / 2 - 1) * std::pow(0.05, 1.0 / numCities));
            updatePheromones(bestTour);
        }

        totalDistance = globalBestDistance;
    }
    const std::vector<int> &getTour() const { return bestTour; }
};

int main(int argc, char *argv[])
{
    std::string filename = (argc > 1) ? argv[1] : "a280.tsp";
    TSPReader reader;
    if (!reader.readFile(filename)) {
        std::cerr << "Failed to read TSP file." << std::endl;
        return 1;
    }
    // reader.printData();

    std::vector<int> bestTour;
    float bestDistance = 0.0, elapsedTime = 0.0;
    std::cout << "Solving TSP...\n";

    auto start = std::chrono::high_resolution_clock::now();

    ACOCUDA solver(reader.getPoints());
    solver.solve();
    bestTour = solver.getTour();
    bestDistance = solver.getDistance();

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    elapsedTime = duration.count();

    // write results to file
    std::string res_fname = "res/" + filename + "_aco_cuda.txt";
    std::ofstream file;
    file.open(res_fname);
    file << "Testcase: " << filename << "\n";
    file << "Time taken: " << elapsedTime << " milliseconds\n";
    file << "Total distance: " << bestDistance << "\n";
    file << "Best tour found:\n";
    for (int city : bestTour) {
        file << city << "\n";
    }

    // print results to console
    std::cout << "Distance: " << bestDistance << "\nSolution found in " << elapsedTime << " milliseconds\n";
    return 0;
}
