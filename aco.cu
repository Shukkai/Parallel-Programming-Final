#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <omp.h>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#define NUM_ANTS 10
#define NUM_ITERATIONS 200
#define ALPHA 1.0
#define BETA 2.0
#define RHO 0.2
#define Q 1.0
#define GRID 2
#define BLOCK 5

struct Point {
    int id;
    float x;
    float y;
};

class TSPReader {
  private:
    std::string name;
    std::string comment;
    std::string type;
    int dimension;
    std::string edge_weight_type;
    std::vector<Point> points;

  public:
    bool readFile(const std::string &filename)
    {
        std::ifstream file(filename);
        if (!file.is_open()) {
            std::cerr << "Error: Could not open file " << filename << std::endl;
            return false;
        }

        std::string line;
        bool reading_coords = false;

        while (std::getline(file, line)) {
            if (reading_coords) {
                std::istringstream iss(line);
                Point p;
                if (iss >> p.id >> p.x >> p.y) {
                    points.push_back(p);
                }
            }
            else {
                if (line.find("NAME") != std::string::npos) {
                    name = line.substr(line.find(":") + 1);
                    name = name.substr(name.find_first_not_of(" \t"));
                }
                else if (line.find("COMMENT") != std::string::npos) {
                    comment = line.substr(line.find(":") + 1);
                }
                else if (line.find("TYPE") != std::string::npos) {
                    type = line.substr(line.find(":") + 1);
                }
                else if (line.find("DIMENSION") != std::string::npos) {
                    dimension = std::stoi(line.substr(line.find(":") + 1));
                }
                else if (line.find("EDGE_WEIGHT_TYPE") != std::string::npos) {
                    edge_weight_type = line.substr(line.find(":") + 1);
                }
                else if (line.find("NODE_COORD_SECTION") != std::string::npos) {
                    reading_coords = true;
                }
            }
        }

        file.close();
        return true;
    }
    void printData() const
    {
        std::cout << "Name: " << name << std::endl;
        std::cout << "Comment: " << comment << std::endl;
        std::cout << "Type: " << type << std::endl;
        std::cout << "Dimension: " << dimension << std::endl;
        std::cout << "Edge Weight Type: " << edge_weight_type << std::endl;
        std::cout << "\nCoordinates:" << std::endl;
        for (const auto &point : points) {
            std::cout << "Point " << point.id << ": (" << point.x << ", " << point.y << ")" << std::endl;
        }
    }
    const std::vector<Point> &getPoints() const { return points; }
};

class TSPSolver {
  protected:
    // data
    std::vector<Point> points;
    float totalDistance;

    // functions
    float calculateDistance(const Point &p1, const Point &p2) const
    {
        float dx = p1.x - p2.x;
        float dy = p1.y - p2.y;
        return std::sqrt(dx * dx + dy * dy);
    }
    float calculateTourDistance(const std::vector<int> &tour) const
    {
        float distance = 0;
        for (size_t i = 0; i < tour.size() - 1; ++i) {
            distance += calculateDistance(points[tour[i]], points[tour[i + 1]]);
        }
        distance += calculateDistance(points[tour.back()], points[tour.front()]);
        return distance;
    }

  public:
    TSPSolver(const std::vector<Point> &pts) : points(pts), totalDistance(std::numeric_limits<float>::infinity()) {}
    float getDistance() { return totalDistance; }
    // void printTour() const;
};

struct Ant {
    int tour[300] = {-1};
    bool visited[300] = {false};
    int numCities;

    Ant(int numCities) : numCities(numCities) {}
    Ant() {}
};

__global__ void setup_kernel(hiprandState *state, unsigned long t)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(t, idx, 0, &state[idx]);
}

__device__ float calculateDistanceCuda(const Point &p1, const Point &p2)
{
    float dx = p1.x - p2.x;
    float dy = p1.y - p2.y;
    return std::sqrt(dx * dx + dy * dy);
}

__device__ int selectNextCityCuda(hiprandState *my_curandstate, int idx, Ant *d_ant, float *d_pheromones,
                                  const Point *points, const int numCities, int current)
{
    float *prob = new float[numCities], total_prob = 0.0;
    memset(prob, 0, numCities * sizeof(float));

    for (int nextCity = 0; nextCity < numCities; nextCity++) {
        if (!d_ant[idx].visited[nextCity]) {
            prob[nextCity] = pow(d_pheromones[current * numCities + nextCity], ALPHA) *
                             pow(1.0 / calculateDistanceCuda(points[current], points[nextCity]), BETA);
            total_prob += prob[nextCity];
        }
    }

    float r = hiprand_uniform(&(my_curandstate[idx])) * total_prob;
    for (int i = 0; i < numCities; i++) {
        if (!d_ant[idx].visited[i]) {
            r -= prob[i];
            if (r <= 0) {
                return i;
            }
        }
    }

    for (int i = 0; i < numCities; i++) {
        if (!d_ant[idx].visited[i]) {
            return i;
        }
    }

    return -1;
}

__global__ void contructSolutionCuda(hiprandState *my_curandstate, Ant *d_ant, float *d_pheromones, const Point *points,
                                     const int numCities)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    d_ant[idx].numCities = numCities;

    float myrandf = hiprand_uniform(&(my_curandstate[idx]));
    myrandf *= ((float)(d_ant[idx].numCities - 1) + 0.999999);
    int current = (int)truncf(myrandf);

    d_ant[idx].tour[0] = current;
    d_ant[idx].visited[current] = true;

    // if (idx == 0)
    // printf("%lf\n", d_pheromones[1]);

    for (int i = 1; i < numCities; i++) {
        int next = selectNextCityCuda(my_curandstate, idx, d_ant, d_pheromones, points, numCities, current);
        d_ant[idx].tour[i] = next;
        d_ant[idx].visited[next] = true;
        current = next;
    }

    // printf("Thread %d, Done\n", idx);
    // printf("%d\n", d_ant[idx].tour[0]);
}

__global__ void restartAnts(Ant *d_ant)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = 0; i < d_ant[idx].numCities; i++) {
        d_ant[idx].tour[i] = -1;
        d_ant[idx].visited[i] = false;
    }
}

class ACOCUDA : public TSPSolver {
  protected:
    // data
    float tau_max, tau_min;
    int numCities;
    // std::vector<std::vector<float>> pheromones;
    std::vector<float> pheromones;
    std::vector<int> bestTour;
    float globalBestDistance = std::numeric_limits<float>::infinity();

    // rng
    std::random_device rd;
    std::mt19937 gen;
    std::uniform_real_distribution<float> uniform_dist;

    // // functions
    // int selectNextCity(int current, const std::vector<bool> &visited)
    // {
    //     std::vector<float> probabilities(numCities, 0.0);
    //     float totalProb = 0.0;

    //     for (int nextCity = 0; nextCity < numCities; nextCity++) {
    //         if (!visited[nextCity]) {
    //             probabilities[nextCity] = std::pow(pheromones[current][nextCity], ALPHA) *
    //                                       std::pow(1.0 / calculateDistance(points[current], points[nextCity]),
    //                                       BETA);
    //             totalProb += probabilities[nextCity];
    //         }
    //     }

    //     float r = uniform_dist(gen) * totalProb;
    //     for (int i = 0; i < numCities; i++) {
    //         if (!visited[i]) {
    //             r -= probabilities[i];
    //             if (r <= 0) {
    //                 return i;
    //             }
    //         }
    //     }

    //     for (int i = 0; i < numCities; i++) {
    //         if (!visited[i]) {
    //             return i;
    //         }
    //     }

    //     return -1;
    // }
    void updatePheromones(const std::vector<int> &allTours)
    {
        // evaporation
        for (int i = 0; i < numCities * numCities; i++) {
            pheromones[i] *= (1.0 - RHO);
        }

        int city1, city2;
        float deposit = 1.0 / globalBestDistance;
        for (int i = 0; i < numCities - 1; i++) {
            city1 = allTours[i];
            city2 = allTours[i + 1];
            pheromones[city1 * numCities + city2] += deposit;
            pheromones[city2 * numCities + city1] += deposit;
            pheromones[city1 * numCities + city2] = std::clamp(pheromones[city1 * numCities + city2], tau_min, tau_max);
            pheromones[city2 * numCities + city1] = std::clamp(pheromones[city2 * numCities + city1], tau_min, tau_max);
        }

        // Add return to start
        city1 = allTours[numCities - 1];
        city2 = allTours[0];
        pheromones[city1 * numCities + city2] += deposit;
        pheromones[city2 * numCities + city1] += deposit;
        pheromones[city1 * numCities + city2] = std::clamp(pheromones[city1 * numCities + city2], tau_min, tau_max);
        pheromones[city2 * numCities + city1] = std::clamp(pheromones[city2 * numCities + city1], tau_min, tau_max);
    }
    // std::vector<int> contructSolution()
    // {
    //     std::vector<bool> visited(numCities, false);
    //     std::vector<int> tour;
    //     tour.reserve(numCities);

    //     int current = std::uniform_int_distribution<int>(0, numCities - 1)(gen);
    //     tour.push_back(current);
    //     visited[current] = true;

    //     for (int i = 1; i < numCities; i++) {
    //         int next = selectNextCity(current, visited);
    //         tour.push_back(next);
    //         visited[next] = true;
    //         current = next;
    //     }

    //     // 2-opt
    //     _2Opt(tour);

    //     return tour;
    // }
    void _2Opt(std::vector<int> &tour)
    {
        while (improve2Opt(tour)) {
            // Continue until no more improvements can be made
        }
    }
    void reverse(std::vector<int> &tour, int start, int end)
    {
        while (start < end) {
            std::swap(tour[start], tour[end]);
            start++;
            end--;
        }
    }
    bool improve2Opt(std::vector<int> &tour)
    {
        int n = tour.size() - 1; // Don't include last city
        bool improved = false;

        for (int i = 0; i < n - 1; i++) {
            for (int j = i + 1; j < n; j++) {
                float beforeDistance = calculateDistance(points[tour[i]], points[tour[i + 1]]) +
                                       calculateDistance(points[tour[j]], points[tour[(j + 1)]]);

                float afterDistance = calculateDistance(points[tour[i]], points[tour[j]]) +
                                      calculateDistance(points[tour[i + 1]], points[tour[(j + 1)]]);

                if (afterDistance < beforeDistance) {
                    reverse(tour, i + 1, j);
                    improved = true;
                }
            }
        }
        return improved;
    }

  public:
    ACOCUDA(const std::vector<Point> &pts) : TSPSolver(pts)
    {
        numCities = points.size();
        pheromones.resize(numCities * numCities, 1.0);
        // gen = std::mt19937(42);
        // uniform_dist = std::uniform_real_distribution<float>(0.0, numCities - 1);
    }

    // Main solving method
    void solve()
    {
        hiprandState *d_state;
        time_t t;
        time(&t);
        hipMalloc(&d_state, NUM_ANTS * sizeof(hiprandState));
        setup_kernel<<<GRID, BLOCK>>>(d_state, t);
        hipDeviceSynchronize();

        Ant *d_ants;
        hipMalloc(&d_ants, NUM_ANTS * sizeof(Ant));

        float *d_pheromones;
        hipMalloc(&d_pheromones, numCities * numCities * sizeof(float));
        // cudaMemcpy(d_pheromones, pheromones.data(), numCities * numCities * sizeof(float), cudaMemcpyHostToDevice);

        Point *d_points;
        hipMalloc(&d_points, numCities * sizeof(Point));
        hipMemcpy(d_points, points.data(), numCities * sizeof(Point), hipMemcpyHostToDevice);

        std::vector<int> tour(numCities);
        for (int iter = 0; iter < NUM_ITERATIONS; iter++) {
            // printf("Iter: %d, %lf\n", iter, pheromones[1]);
            hipMemcpy(d_pheromones, pheromones.data(), numCities * numCities * sizeof(float), hipMemcpyHostToDevice);

            contructSolutionCuda<<<GRID, BLOCK>>>(d_state, d_ants, d_pheromones, d_points, numCities);
            hipDeviceSynchronize();

            for (int ant = 0; ant < NUM_ANTS; ant++) {
                hipMemcpy(tour.data(), d_ants[ant].tour, numCities * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(pheromones.data(), d_pheromones, numCities * numCities * sizeof(float),
                           hipMemcpyDeviceToHost);

                _2Opt(tour);
                float tourLength = calculateTourDistance(tour);

                if (tourLength < globalBestDistance) {
                    bestTour = tour;
                    globalBestDistance = tourLength;
                    // std::cout << "Iteration " << iter << ": Best tour length = " << globalBestDistance <<
                    // std::endl;
                }
            }

            // printf("Iter: %d, Tour Length: %f\n", iter, globalBestDistance);

            tau_max = 1.0 / (RHO * globalBestDistance);
            tau_min = tau_max * (1.0 - std::pow(0.05, 1.0 / numCities)) /
                      ((numCities / 2 - 1) * std::pow(0.05, 1.0 / numCities));
            updatePheromones(bestTour);

            restartAnts<<<GRID, BLOCK>>>(d_ants);
        }

        totalDistance = globalBestDistance;
    }
    const std::vector<int> &getTour() const { return bestTour; }
};

int main(int argc, char *argv[])
{
    std::string filename = (argc > 0) ? argv[1] : "a280.tsp";
    TSPReader reader;
    if (!reader.readFile(filename)) {
        std::cerr << "Failed to read TSP file." << std::endl;
        return 1;
    }
    // reader.printData();

    std::vector<int> bestTour;
    float bestDistance = 0.0, elapsedTime = 0.0;
    std::cout << "Solving TSP...\n";

    auto start = std::chrono::high_resolution_clock::now();

    ACOCUDA solver(reader.getPoints());
    solver.solve();
    bestTour = solver.getTour();
    bestDistance = solver.getDistance();

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    elapsedTime = duration.count();

    // write results to file
    std::string res_fname = "res/aco_cuda.txt";
    std::ofstream file;
    file.open(res_fname);
    file << "Testcase: " << filename << "\n";
    file << "Time taken: " << elapsedTime << " milliseconds\n";
    file << "Total distance: " << bestDistance << "\n";
    file << "Best tour found:\n";
    for (int city : bestTour) {
        file << city << "\n";
    }

    // print results to console
    std::cout << "Distance: " << bestDistance << "\nSolution found in " << elapsedTime << " milliseconds\n";
    return 0;
}